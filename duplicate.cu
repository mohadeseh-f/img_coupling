#include "hip/hip_runtime.h"
#include"duplicate.h"

__global__ void duplication_kernel(int *output,int*origin_data, int *data, int size){
	
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int j=tid%size;
	int i=(tid-j)/size;		
			for ( int k = 0; k < img_size; k++){
				// printf("img_in[i]: %d\n",  img_in[i]);
				// printf("img_in[j]:%d\n",  img_in[j]);
				int diff = abs (img_in[i*img_size +k] - img_in[j*img_size +k]);
				if (diff == 0)
					num_of_one++;
				
			}
			percent[i * img_size + j]= (num_of_one);
			// int darsad = (num_of_one*100)/img_size;
			// printf("darsad tashabohe axe %d ba axe %d hast %d \n", counter , counter+repeat+1 ,darsad);
			//printf("darsad tashabohe axe %d ba axe %d hast %d \n", i , j ,num_of_one);
		

	
}
void sequential_duplicate(int *percent,int *img_in, int img_size){
// for(int p = 0 ; p< img_size*img_num; p++){
// 	printf("%d\t", img_in[p] );
// }	

	for(int i = 0; i < img_num; i++){
		for(int j = 0  ; j < img_num ; j++){
			int num_of_one=0;
			
			for ( int k = 0; k < img_size; k++){
				// printf("img_in[i]: %d\n",  img_in[i]);
				// printf("img_in[j]:%d\n",  img_in[j]);
				int diff = abs (img_in[i*img_size +k] - img_in[j*img_size +k]);
				if (diff == 0)
					num_of_one++;
				
			}
			percent[i * img_size + j]= (num_of_one);
			// int darsad = (num_of_one*100)/img_size;
			// printf("darsad tashabohe axe %d ba axe %d hast %d \n", counter , counter+repeat+1 ,darsad);
			printf("darsad tashabohe axe %d ba axe %d hast %d \n", i , j ,num_of_one);
		}

	}
	return;
}

int main(int argc, char *argv[]){

	double elapsed_time;
	int block_size_x, grid_size_x;
	int input_size;
	int output_size;
	int *input_h, *output_h, *device_output_h;
	int *input_d,*origin_input_d, *output_d;
	int stream_count = img_num*(img_num+1)/2;
	// int work_per_thread;

	// if(argc != 2){
	// 	printf("Correct way to execute this program is:\n");
	// 	printf("./blur block_size_x block_size_y stream_count\n");
	// 	printf("For example:\n./blur 16 16 \n");
	// 	return 1;
	// }

	
	input_size = IMAGE_SIZE_X * IMAGE_SIZE_Y;
	output_size = img_num * img_num;

	// har 2 ta ax dar yek block bashand 
	block_size_x = 2*input_size;
	

//	hipStream_t* streams = (hipStream_t *)malloc(sizeof(hipStream_t) * STREAM_NUMBERS);

	// for(int i = 0; i < STREAM_NUMBERS; i++){
	// 	hipStreamCreate(&streams[i]);
	// }

	// Initialize data on Host
	int count;
	initialize_data_random_cudaMallocHost(&input_h, input_size*img_num);
	
	// Initialize data on Host
	
	//initialize_data_zero(&output_h, output_size);
	initialize_data_zero_cudaMallocHost(&output_h, output_size);
	//initialize_data_zero_cudaMallocHost(&device_output_h, output_size);
	initialize_data_zero_cudaMallocHost(&output_device_h, output_size);
	// Initialize data on Device
	CUDA_CHECK_RETURN(hipMalloc((void **)&input_d, sizeof(int)*input_size*img_num));
	CUDA_CHECK_RETURN(hipMalloc((void **)&origin_input_d, sizeof(int)*output_size));

	CUDA_CHECK_RETURN(hipMalloc((void **)&output_d, sizeof(int)*output_size));
	
	// Perform GPU Warm-up
	// CUDA_CHECK_RETURN(hipMemcpyAsync(input_d, input_h, sizeof(int), hipMemcpyHostToDevice, streams[0]));

	// Sequential blur operation
	// 
	set_clock();

	sequential_duplicate(output_h,input_h, input_size);


    elapsed_time = get_elapsed_time();

	printf("->sequential duplication time: %.4fms\n", elapsed_time / 1000);

	// CUDA Parallel duplication


	set_clock();


	// int stream_size = 2*input_size;
	// int stream_bytes = stream_size * sizeof(input_d[0]);

	grid_size_x =  img_num*(img_num);
	dim3 grid_dime(1, 1, 1);
	dim3 block_dime(grid_size_x, 1, 1);
	

	//stream count = tedade dafAti k in 2 halghe tekrar mishavand yani dar vaghe ma ruye stream count darim loop mizanim vali chon b offset 
	// niyaz darim majburim an ra b 2 hakgheye mojaza taghsim konim
	// for(int counter  = 0; counter < img_num; counter ++){
	// 	int origin_offset = counter*img_size;
		
	// 	for (int repeat=0 ; repeat< img_num - counter; repeat++){
			
			//int offset = img_size*(repeat+counter+1);
			hipMemcpy(&input_d, &input_h, input_size*img_num, hipMemcpyHostToDevice);
			//hipMemcpy(&input_d[offset], &input_h[offset], stream_bytes/2, hipMemcpyHostToDevice, streams[ repeat% STREAM_NUMBERS]);
		
			duplication_kernel<<< grid_dime, block_dime>>>(&output_d, &input_h,input_size);

			 hipMemcpyAsync(&output_device_h, &output_d, output_size, hipMemcpyDeviceToHost);
			// offset += stream_size;
	// 	}
	// }

	CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());
	
    elapsed_time = get_elapsed_time();

    printf("-> CUDA duplication time: %.4fms\n", elapsed_time / 1000);

 //    #ifdef  TEST
     validate(output_h, device_output_h, img_num*img_num);
 //    #endif

	// for (int i = 0; i < STREAM_NUMBERS; i++){
 //        hipStreamDestroy(streams[i]);
 //    }
	// //free(data_h);
	// CUDA_CHECK_RETURN(hipHostFree(data_h));
	// free(output_h);
	// free(streams);
	// //free(device_output_h);
	// CUDA_CHECK_RETURN(hipHostFree(device_output_h));

	// CUDA_CHECK_RETURN(hipFree(output_d));
	// CUDA_CHECK_RETURN(hipFree(data_d));

	return 0;
}

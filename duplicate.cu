#include "hip/hip_runtime.h"

/* this is project of GPU course in shiraz university  project obout finding duplicated pictuers 
*/
#include"duplicate.h"


 // this function used for kernel in GPU

__global__ void duplication_kernel(int *output, int *data, int size){
	
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int j=tid%size;
	int i=(tid-j)/size;		
			int num_of_one=0;
			for ( int k = 0; k < size; k++){
				int diff = abs (data[i*size +k] - data[j*size +k]);
				if (diff == 0)
					num_of_one++;
				
			}
			output[i * size + j]= (num_of_one*100)/(RANDOM_NUMBER_MAX*size *size);
	
}


// this function used sequential implimentation of duplicaded pictures and calcuate time
void sequential_duplicate(int *percent,int *img_in, int img_size){
	
	for(int i = 0; i < img_num; i++){
		for(int j = 0  ; j < img_num ; j++){
			int num_of_one=0;
			
			for ( int k = 0; k < img_size; k++){
				int diff = abs (img_in[i*img_size +k] - img_in[j*img_size +k]);
				if (diff == 0)
					num_of_one++;
				
			}
			percent[i * img_size + j]= (num_of_one*100)/(RANDOM_NUMBER_MAX*img_size *img_size);
			printf("darsad tashabohe axe %d ba axe %d hast %d \n", i , j ,num_of_one);
		}

	}
	return;
}

int main(int argc, char *argv[]){

	double elapsed_time;
	int block_size_x, grid_size_x;
	int input_size;
	int output_size;
	int *input_h, *output_h, *output_device_h;
	int *input_d,*origin_input_d, *output_d;
	int stream_count = img_num*(img_num+1)/2;

	
	input_size = IMAGE_SIZE_X * IMAGE_SIZE_Y;
	output_size = img_num * img_num;
	block_size_x = 2*input_size;
	

	int count;
	initialize_data_random_cudaMallocHost(&input_h, input_size*img_num);
	
	// Initialize data on Host
	
	//initialize_data_zero(&output_h, output_size);
	initialize_data_zero_cudaMallocHost(&output_h, output_size);
	//initialize_data_zero_cudaMallocHost(&device_output_h, output_size);
	initialize_data_zero_cudaMallocHost(&output_device_h, output_size);
	// Initialize data on Device
	CUDA_CHECK_RETURN(hipMalloc((void **)&input_d, sizeof(int)*input_size*img_num));
	CUDA_CHECK_RETURN(hipMalloc((void **)&origin_input_d, sizeof(int)*output_size));

	CUDA_CHECK_RETURN(hipMalloc((void **)&output_d, sizeof(int)*output_size));
	

	// Sequential opration
	// 
	set_clock();

	sequential_duplicate(output_h,input_h, input_size);


    elapsed_time = get_elapsed_time();

	printf("->sequential duplication time: %.4fms\n", elapsed_time / 1000);

	// CUDA Parallel duplication


	set_clock();


	// this part calculate gride size and block size for GPU
	grid_size_x =  img_num*(img_num);
	dim3 grid_dime(1, 1, 1);
	dim3 block_dime(grid_size_x, 1, 1);
	

	
	hipMemcpy(&input_d, &input_h, input_size*img_num, hipMemcpyHostToDevice);
			
	duplication_kernel<<< grid_dime, block_dime>>>(output_d, input_h, input_size);

	hipMemcpy(&output_device_h, &output_d, output_size, hipMemcpyDeviceToHost);
		

	CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());
	
    elapsed_time = get_elapsed_time();

    printf("-> CUDA duplication time: %.4fms\n", elapsed_time / 1000);

     validate(output_h, output_device_h, img_num*img_num);
 
 	 // fre allocated memories 
	free(input_d);
	free(output_d);
	
	CUDA_CHECK_RETURN(hipHostFree(output_device_h));

	CUDA_CHECK_RETURN(hipFree(output_h));
	CUDA_CHECK_RETURN(hipFree(input_h));

	return 0;
}
